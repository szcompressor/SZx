#include "cuSZx_entry.h"
#include "timingGPU.h"

TimingGPU timer_GPU;

unsigned char* cuSZx_fast_compress_args_unpredictable_blocked_float(float *oriData, size_t *outSize, float absErrBound, size_t nbEle, int blockSize, unsigned char *test)
{

	float* d_oriData;
    hipMalloc((void**)&d_oriData, sizeof(float)*nbEle); 
    hipMemcpy(d_oriData, oriData, sizeof(float)*nbEle, hipMemcpyHostToDevice); 

	size_t nbBlocks = nbEle/blockSize;
	size_t remainCount = nbEle%blockSize;
	size_t actualNBBlocks = remainCount==0 ? nbBlocks : nbBlocks+1;

    size_t ncBytes = blockSize/4;
    //ncBytes = (blockSize+1)%4==0 ? ncBytes : ncBytes+1; //Bytes to store one non-constant block data.
    size_t mSize = 1+sizeof(float)+1+ncBytes+sizeof(unsigned int); //Number of bytes for each data block's metadata.
    size_t msz = mSize * nbBlocks * sizeof(unsigned char);
    size_t mbsz = sizeof(float) * nbEle * sizeof(unsigned char);

    unsigned char *meta = (unsigned char*)malloc(msz);
    unsigned char *midBytes = (unsigned char*)malloc(mbsz);

	unsigned char* d_meta;
	unsigned char* d_midBytes;
    checkCudaErrors(hipMalloc((void**)&d_meta, msz)); 
    //checkCudaErrors(hipMemcpy(d_meta, meta, msz, hipMemcpyHostToDevice)); 
    checkCudaErrors(hipMemset(d_meta, 0, msz));
    checkCudaErrors(hipMalloc((void**)&d_midBytes, mbsz)); 
    //checkCudaErrors(hipMemset(d_midBytes, 0, mbsz));
    //hipMemcpy(dresults, results, sizeof(unsigned char)*reSize*nbBlocks, hipMemcpyHostToDevice); 
    for (int i=0; i<nbBlocks; i++) 
        printf ("test:%d\n", i);

    //timer_GPU.StartCounter();
    //dim3 dimBlock(32, blockSize/32);
    //dim3 dimGrid(512, 1);
    //const int sMemsize = 10 * dimBlock.y * sizeof(double);
    ////compress_float<<<dimGrid, dimBlock, sMemsize>>>(d_oriData, d_meta, d_midBytes, absErrBound, blockSize, nbBlocks, mSize);
    //hipError_t err = hipGetLastError();        // Get error code
    //printf("CUDA Error: %s\n", hipGetErrorString(err));
    //checkCudaErrors(hipMemcpy(meta, d_meta, msz, hipMemcpyDeviceToHost)); 

    //for (int i=0; i<nbBlocks; i++){ 
    //    //if (meta[i*mSize]!=test[i]) 
    //    //printf("state %d : %u\n", i, test[i]);
    //    printf ("test:%d\n", nbBlocks);
    //}
}
