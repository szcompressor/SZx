#include "cuSZx_entry.h"
#include "defines.h"
#include "ByteToolkit.h"
#include "TypeManager.h"
#include "timingGPU.h"

TimingGPU timer_GPU;
void bin(unsigned n)
{
    unsigned i;
    for (i = 1 << 31; i > 0; i = i / 2)
        (n & i) ? printf("1") : printf("0");
}

int _post_proc(float *oriData, unsigned char *meta, short *offsets, unsigned char *midBytes, unsigned char *outBytes, size_t nbEle, int blockSize)
{
    int out_size = 0;
    size_t nbConstantBlocks = 0;
    size_t nbBlocks = nbEle/blockSize;
    size_t ncBytes = blockSize/4;
    size_t mSize = sizeof(float)+1+ncBytes; //Number of bytes for each data block's metadata.
    out_size += 5+sizeof(size_t)+sizeof(float)*nbBlocks;
    if (nbBlocks%8==0)
        out_size += nbBlocks/8;
    else
        out_size += nbBlocks/8+1;
    for (int i=0; i<nbBlocks; i++){
        if (meta[i]==0) nbConstantBlocks++;
        else out_size += 1+(blockSize/4)+offsets[i];
    }
    out_size += (nbBlocks-nbConstantBlocks)*sizeof(short)+(nbEle%blockSize)*sizeof(float);

    //outBytes = (unsigned char*)malloc(out_size);
	unsigned char* r = outBytes; 
	r[0] = SZ_VER_MAJOR;
	r[1] = SZ_VER_MINOR;
	r[2] = SZ_VER_SUPERFAST;
	r[3] = 0; // indicates this is not a random access version
	r[4] = (unsigned char)blockSize;
	r=r+5; //1 byte
	sizeToBytes(r, nbConstantBlocks);
	r += sizeof(size_t); 
	r += convertIntArray2ByteArray_fast_1b_args(meta, nbBlocks, r);
    memcpy(r, oriData+nbBlocks*blockSize, (nbEle%blockSize)*sizeof(float));
    r += (nbEle%blockSize)*sizeof(float);
    unsigned char* c = r;
    unsigned char* o = c+nbConstantBlocks*sizeof(float);
    unsigned char* nc = o+(nbBlocks-nbConstantBlocks)*sizeof(short);
    for (int i=0; i<nbBlocks; i++){
        
        if (meta[i]==0){
            memcpy(c, meta+(nbBlocks+i*mSize), sizeof(float));
            c += sizeof(float);
        }else{
            shortToBytes(o, offsets[i]);
            o += sizeof(short);
            memcpy(nc, meta+(nbBlocks+i*mSize), mSize);
            nc += mSize; 
            memcpy(nc, midBytes+(i*blockSize*sizeof(float)), offsets[i]);
            nc += offsets[i];
        } 
    }

    return out_size;
}

unsigned char* cuSZx_fast_compress_args_unpredictable_blocked_float(float *oriData, size_t *outSize, float absErrBound, size_t nbEle, int blockSize, unsigned char *test)
{

	float* d_oriData;
    hipMalloc((void**)&d_oriData, sizeof(float)*nbEle); 
    hipMemcpy(d_oriData, oriData, sizeof(float)*nbEle, hipMemcpyHostToDevice); 

	size_t nbBlocks = nbEle/blockSize;
	size_t remainCount = nbEle%blockSize;
	size_t actualNBBlocks = remainCount==0 ? nbBlocks : nbBlocks+1;

    size_t ncBytes = blockSize/4;
    //ncBytes = (blockSize+1)%4==0 ? ncBytes : ncBytes+1; //Bytes to store one non-constant block data.
    size_t mSize = sizeof(float)+1+ncBytes; //Number of bytes for each data block's metadata.
    size_t msz = (1+mSize) * nbBlocks * sizeof(unsigned char);
    size_t mbsz = sizeof(float) * nbEle * sizeof(unsigned char);

    unsigned char *meta = (unsigned char*)malloc(msz);
    short *offsets = (short*)malloc(nbBlocks*sizeof(short));
    unsigned char *midBytes = (unsigned char*)malloc(mbsz);
    int *dtest = (int*)malloc(nbBlocks * sizeof(int));

	unsigned char* d_meta;
	unsigned char* d_midBytes;
	short* d_offsets;
    int *d_test;
    checkCudaErrors(hipMalloc((void**)&d_meta, msz)); 
    //checkCudaErrors(hipMemcpy(d_meta, meta, msz, hipMemcpyHostToDevice)); 
    checkCudaErrors(hipMemset(d_meta, 0, msz));
    checkCudaErrors(hipMalloc((void**)&d_offsets, nbBlocks*sizeof(short))); 
    checkCudaErrors(hipMemset(d_offsets, 0, nbBlocks*sizeof(short)));
    checkCudaErrors(hipMalloc((void**)&d_midBytes, mbsz)); 
    checkCudaErrors(hipMemset(d_midBytes, 0, mbsz));
    //hipMemcpy(dresults, results, sizeof(unsigned char)*reSize*nbBlocks, hipMemcpyHostToDevice); 
    checkCudaErrors(hipMalloc((void**)&d_test, nbBlocks * sizeof(int))); 
    checkCudaErrors(hipMemset(d_test, 0, nbBlocks * sizeof(int)));

    //timer_GPU.StartCounter();
    dim3 dimBlock(32, blockSize/32);
    dim3 dimGrid(512, 1);
    const int sMemsize = blockSize * sizeof(float) + dimBlock.y * sizeof(int);
    compress_float<<<dimGrid, dimBlock, sMemsize>>>(d_oriData, d_meta, d_offsets, d_midBytes, absErrBound, blockSize, nbBlocks, mSize, d_test);
    hipError_t err = hipGetLastError();        // Get error code
    printf("CUDA Error: %s\n", hipGetErrorString(err));
    checkCudaErrors(hipMemcpy(meta, d_meta, msz, hipMemcpyDeviceToHost)); 
    checkCudaErrors(hipMemcpy(offsets, d_offsets, nbBlocks*sizeof(short), hipMemcpyDeviceToHost)); 
    checkCudaErrors(hipMemcpy(midBytes, d_midBytes, mbsz, hipMemcpyDeviceToHost)); 
    checkCudaErrors(hipMemcpy(dtest, d_test, nbBlocks * sizeof(int), hipMemcpyDeviceToHost)); 

    size_t maxPreservedBufferSize = sizeof(float)*nbEle;
    unsigned char* outBytes = (unsigned char*)malloc(maxPreservedBufferSize);
    memset(outBytes, 0, maxPreservedBufferSize);

    *outSize = _post_proc(oriData, meta, offsets, midBytes, outBytes, nbEle, blockSize);
    printf("size %u\n", outBytes[4]);

    //for (int i=0; i<nbBlocks; i++){ 
    //    if (dtest[i]!=test[i]){
    //        bin(dtest[i]);
    //        printf("state %d : %i, %i\n", i, test[i], dtest[i]);
    //    } 
    //}
    for (int i=0; i<sizeof(float) * nbEle; i++){ 
        if (midBytes[i]!=test[i]){
            bin(midBytes[i]);
            printf("state %d : %u, %u\n", i, test[i], midBytes[i]);
        } 
    }
    free(meta);
    free(offsets);
    free(midBytes);
    free(dtest);
    checkCudaErrors(hipFree(d_meta));
    checkCudaErrors(hipFree(d_offsets));
    checkCudaErrors(hipFree(d_midBytes));
    checkCudaErrors(hipFree(d_test));
    return outBytes;
}
