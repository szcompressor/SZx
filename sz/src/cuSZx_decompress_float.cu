#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuSZx_decompress_float.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

__device__ int _deshfl_scan(int lznum, int *sums)
{
    // Below is the basic structure of using a shfl instruction
    // for a scan.
    // Record "value" as a variable - we accumulate it along the way
    int value = lznum;

    // Now accumulate in log steps up the chain
    // compute sums, with another thread's value who is
    // distance delta away (i).  Note
    // those threads where the thread 'i' away would have
    // been out of bounds of the warp are unaffected.  This
    // creates the scan sum.

#pragma unroll
    for (int i = 1; i <= warpSize; i *= 2) {
        unsigned int mask = 0xffffffff;
        int n = __shfl_up_sync(mask, value, i);

        if (threadIdx.x >= i) value += n;
                      
    }

    // value now holds the scan value for the individual thread
    // next sum the largest values for each warp

    // write the sum of the warp to smem
    if (threadIdx.x == warpSize - 1) {
        sums[threadIdx.y] = value;
    }
    __syncthreads();

    //
    // scan sum the warp sums
    // the same shfl scan operation, but performed on warp sums
    //
    if (threadIdx.y == 0 && threadIdx.x < blockDim.y) {
        int warp_sum = sums[threadIdx.x];

        int mask = (1 << blockDim.y) - 1;
        for (int i = 1; i <= blockDim.y; i *= 2) {
            //int n = __shfl_up_sync(mask, warp_sum, i, blockDim.y);
            int n = __shfl_up_sync(mask, warp_sum, i);
            if (threadIdx.x >= i) warp_sum += n;
        }

        sums[threadIdx.x] = warp_sum;
    }
    __syncthreads();

    // perform a uniform add across warps in the block
    // read neighbouring warp's sum and add it to threads value
    int blockSum = 0;
    if (threadIdx.y > 0) {
        blockSum = sums[threadIdx.y - 1];
    }
    value += blockSum;

    return value;
}

__device__ int _compareByte(int pre, int cur, int reqBytesLength)
{
        //if ((cur&0xff)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,threadIdx.x,threadIdx.y,cur&0xff);
        //if ((cur&0xff00)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,threadIdx.x,threadIdx.y,cur&0xff00);
        //if ((cur&0xff0000)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,threadIdx.x,threadIdx.y,cur&0xff0000);
        //if ((cur&0xff000000)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,threadIdx.x,threadIdx.y,cur&0xff000000);
    if (reqBytesLength == 2)
    {
        if ((pre&0x0000ff00) > (cur&0x0000ff00)){
            cur &= 0x000000ff;
            cur |= (pre & 0x0000ff00);
        }
        if ((pre&0x000000ff) > (cur&0x000000ff)){
            cur &= 0x0000ff00;
            cur |= (pre & 0x000000ff);
        }
    }else if (reqBytesLength == 3)
    {
        if ((pre&0x00ff0000) > (cur&0x00ff0000)){
            cur &= 0x0000ffff;
            cur |= (pre & 0x00ff0000);
        }
        if ((pre&0x0000ff00) > (cur&0x0000ff00)){
            cur &= 0x00ff00ff;
            cur |= (pre & 0x0000ff00);
        }
        if ((pre&0x000000ff) > (cur&0x000000ff)){
            cur &= 0x00ffff00;
            cur |= (pre & 0x000000ff);
        }
    }else if (reqBytesLength == 1)
    {
        if (pre > cur)
            cur = pre;
    }else if (reqBytesLength == 4)
    {
        if ((pre&0xff000000) > (cur&0xff000000)){
            cur &= 0x00ffffff;
            cur |= (pre & 0xff000000);
        }
        if ((pre&0x00ff0000) > (cur&0x00ff0000)){
            cur &= 0xff00ffff;
            cur |= (pre & 0x00ff0000);
        }
        if ((pre&0x0000ff00) > (cur&0x0000ff00)){
            cur &= 0xffff00ff;
            cur |= (pre & 0x0000ff00);
        }
        if ((pre&0x000000ff) > (cur&0x000000ff)){
            cur &= 0xffffff00;
            cur |= (pre & 0x000000ff);
        }
    }
    return cur;
}

__device__ int _retrieve_leading(int pos, int reqBytesLength, int* sums)
{
        //if ((pos&0xff)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,threadIdx.x,threadIdx.y,pos&0x000000ff);
        //if ((pos&0xff00)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,threadIdx.x,threadIdx.y,pos&0x0000ff00);
        //if ((pos&0x00ff0000)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,threadIdx.x,threadIdx.y,pos&0x00ff0000);
        //if ((pos&0xff000000)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,threadIdx.x,threadIdx.y,pos&0xff000000);
#pragma unroll
    for (int i = 1; i <= warpSize; i *= 2) {
        unsigned int mask = 0xffffffff;
        int n = __shfl_up_sync(mask, pos, i);
        if (threadIdx.x >= i)
            pos = _compareByte(n, pos, reqBytesLength);
    }

    if (threadIdx.x == warpSize - 1)
        sums[threadIdx.y] = pos;
    __syncthreads();

    if (threadIdx.y == 0 && threadIdx.x < blockDim.y) {
        int warp_pos = sums[threadIdx.x];

        int mask = (1 << blockDim.y) - 1;
        for (int i = 1; i <= blockDim.y; i *= 2) {
            int n = __shfl_up_sync(mask, warp_pos, i);
            if (threadIdx.x >= i)
                warp_pos = _compareByte(n, warp_pos, reqBytesLength);
        }

        sums[threadIdx.x] = warp_pos;
    }
    __syncthreads();

    if (threadIdx.y > 0) {
        int block_pos = sums[threadIdx.y - 1];
        pos = _compareByte(block_pos, pos, reqBytesLength);
    }

    return pos;
}

__global__ void decompress_float(unsigned char *data, int bs, size_t nc, size_t mSize, int *test) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int tid = tidy*warpSize+tidx;
    int bid = blockIdx.x;

    float newData, medianValue;
    unsigned mask;
    unsigned char leadingNum;
    extern __shared__ float shared[];
    float* value = shared;
    int* ivalue = (int*)shared;
    uchar4* c4value = (uchar4*)shared;
    unsigned char* cvalue = (unsigned char*)shared;
    int* sums = &ivalue[bs];
    int reqLength;
    uchar4* uc4bytes = (uchar4*)data;
    float* fbytes = (float*)data;
	int reqBytesLength;
	int rightShiftBits;


    bool bi = false;
    if (bid==73) bi=true;
    for (int b=bid; b<nc; b+=gridDim.x){
        c4value[tid] = uc4bytes[b*bs+tid];
        __syncthreads();                  
        medianValue = value[0];
        reqLength = (int)cvalue[4];
        //if (b<2&&tidx==0&&tidy==0) printf("sss%d:%d\n",b, reqLength);
        if (reqLength%8 != 0)
        {
            reqBytesLength = reqLength/8+1;		
            rightShiftBits = 8 - reqLength%8;
        }else{
            reqBytesLength = reqLength/8;		
            rightShiftBits = 0;
        }
        leadingNum = cvalue[5+(tid>>2)];
        leadingNum = (leadingNum >> (6-((tid&0x03)<<1))) & 0x03;
        int midByte_size = reqBytesLength - leadingNum;
        int midByte_sum = _deshfl_scan(midByte_size, sums);

        uchar4 tmp;
        tmp.x = 0;
        tmp.y = 0;
        tmp.z = 0;
        tmp.w = 0;
        int pos = 0;
        if (reqBytesLength == 2)
        {
            if (midByte_size == 1){
                tmp.z = cvalue[mSize+midByte_sum-1]; 
                pos |= tid<<8;
                //if (bi==true) printf("%i:%i:%i:%u\n", blockIdx.x, threadIdx.x, threadIdx.y, cur_cvalue.z);
            }else if (midByte_size == 2){
                tmp.w = cvalue[mSize+midByte_sum-1]; 
                //if (bi==true) printf("%i:%i:%i:%u\n", blockIdx.x, threadIdx.x, threadIdx.y, cur_cvalue.z);
                tmp.z = cvalue[mSize+midByte_sum-2];
                pos |= tid;
                pos |= tid<<8;
                //if (bi==true) printf("%i:%i:%i:%u\n", blockIdx.x, threadIdx.x, threadIdx.y, cur_cvalue.w);
            }
        }else if (reqBytesLength == 3)
        {
            if (midByte_size == 1){
                tmp.y = cvalue[mSize+midByte_sum-1]; 
                pos |= tid<<16;
            }else if (midByte_size == 2){
                tmp.z = cvalue[mSize+midByte_sum-1]; 
                tmp.y = cvalue[mSize+midByte_sum-2]; 
                pos |= tid<<8;
                pos |= tid<<16;
            }else if (midByte_size == 3){
                tmp.w = cvalue[mSize+midByte_sum-1]; 
                tmp.z = cvalue[mSize+midByte_sum-2]; 
                tmp.y = cvalue[mSize+midByte_sum-3]; 
                pos |= tid;
                pos |= tid<<8;
                pos |= tid<<16;
            }
        }else if (reqBytesLength == 1)
        {
            if (midByte_size == 1)
                tmp.w = cvalue[mSize+midByte_sum-1]; 
                pos |= tid;
        }else if (reqBytesLength == 4)
        {
            if (midByte_size == 1){
                tmp.x = cvalue[mSize+midByte_sum-1]; 
                pos |= tid<<24;
            }else if (midByte_size == 2){
                tmp.y = cvalue[mSize+midByte_sum-1]; 
                tmp.x = cvalue[mSize+midByte_sum-2]; 
                pos |= tid<<16;
                pos |= tid<<24;
            }else if (midByte_size == 3){
                tmp.z = cvalue[mSize+midByte_sum-1]; 
                tmp.y = cvalue[mSize+midByte_sum-2]; 
                tmp.x = cvalue[mSize+midByte_sum-3]; 
                pos |= tid<<8;
                pos |= tid<<16;
                pos |= tid<<24;
            }else if (midByte_size == 4){
                tmp.w = cvalue[mSize+midByte_sum-1]; 
                tmp.z = cvalue[mSize+midByte_sum-2]; 
                tmp.y = cvalue[mSize+midByte_sum-3]; 
                tmp.x = cvalue[mSize+midByte_sum-4]; 
                pos |= tid;
                pos |= tid<<8;
                pos |= tid<<16;
                pos |= tid<<24;
            }
        }
        __syncthreads();                  
        c4value[tid] = tmp;
        __syncthreads();                  

        pos = _retrieve_leading(pos, reqBytesLength, sums);
        //if ((pos&0xff)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,tidx,tidy,pos&0xff);
        //if ((pos&0xff00)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,tidx,tidy,pos&0xff00);
        //if ((pos&0xff0000)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,tidx,tidy,pos&0xff0000);
        //if ((pos&0xff000000)>63)
        //    printf("sss%d:%d,%d,%u\n",reqBytesLength,tidx,tidy,pos&0xff000000);

        if (leadingNum == 2){
            tmp.w = c4value[pos&0xff].w; 
            tmp.z = c4value[(pos>>8)&0xff].z;
        }else if (leadingNum == 3){
            tmp.w = c4value[pos&0xff].w; 
            tmp.z = c4value[(pos>>8)&0xff].z;
            tmp.y = c4value[(pos>>16)&0xff].y; 
        }else if (leadingNum == 1){
            tmp.w = c4value[pos&0xff].w; 
        }else if (leadingNum == 4){
            tmp.w = c4value[pos&0xff].w; 
            tmp.z = c4value[(pos>>8)&0xff].z;
            tmp.y = c4value[(pos>>16)&0xff].y; 
            tmp.x = c4value[pos>>24].x; 
        }
        __syncthreads();                  
        c4value[tid] = tmp;
        ivalue[tid] = ivalue[tid] << rightShiftBits;
        __syncthreads();                  

        newData = value[tid] + medianValue;
        if (b<1) printf("sss%d:%d,%d,%f\n",reqBytesLength,tidx,tidy,newData);

        //if (tidx<2 && tidy==0)
        //    ivalue[tidx] = offsets[b+tidx];
        //__syncthreads();                  
        //obase = ivalue[0];
        //osize = (ivalue[1]-ivalue[0])%4==0 ? (ivalue[1]-ivalue[0])/4 : (ivalue[1]-ivalue[0])/4+1;
        //int* uc4bytes = (int*)(ncBytes+obase); 
        //__syncthreads();                  
        //if (b==0&&tidx==0&&tidy==0) printf("test:%d\n", osize);
        //for (int t=tid; t<osize; t+=blockDim.y*blockDim.x){
        //    int tmp = uc4bytes[t];
        //    ivalue[t] = tmp;
        //    if (b==0) printf("sss:%u\n", t);
        //}
        //__syncthreads();                  
        //medianValue = value[0];
        //if (b==0&&tidx==0&&tidy==0)
        //    printf("median:%f\n", medianValue);


        
        //data = oriData[b*bs+tidy*warpSize+tidx];
        //float Min = data;
        //float Max = data;

        //for (int offset = warpSize/2; offset > 0; offset /= 2) 
        //{
        //    Min = min(Min, __shfl_xor_sync(FULL_MASK, Min, offset));
        //    Max = max(Max, __shfl_xor_sync(FULL_MASK, Max, offset));
        //}
        //if (tidx==0){
        //    value[tidy] = Min;
        //    value[blockDim.y+tidy] = Max;
        //}
        //__syncthreads();                  

        //if (tidy==0){
        //    if (tidx < blockDim.y){
        //        Min = value[tidx];
        //        Max = value[blockDim.y+tidx];
        //    }

        //    mask = __ballot_sync(FULL_MASK, tidx < blockDim.y);
        //    for (int offset = blockDim.y/2; offset > 0; offset /= 2) 
        //    {
        //        Min = min(Min, __shfl_xor_sync(mask, Min, offset));
        //        Max = max(Max, __shfl_xor_sync(mask, Max, offset));
        //    }
        //    
        //    if (tidx==0){
        //        radius = (Max - Min)/2;
        //        value[0] = radius;
        //        value[1] = Min + radius;
        //        value[2] = absErrBound;
        //    }
        //}
        //__syncthreads();                  

        //radius = value[0];
        //medianValue = value[1];
        //state = radius <= absErrBound ? 0 : 1;
        //if (tidx==0){
        //    meta[b] = state;
        //    meta[nb+b*mSize] = cvalue[1].x;
        //    meta[nb+b*mSize+1] = cvalue[1].y;
        //    meta[nb+b*mSize+2] = cvalue[1].z;
        //    meta[nb+b*mSize+3] = cvalue[1].w;
        //} 
        ////if (tidx==0) test[b] = ivalue[0];
        //__syncthreads();                  

        //if (state==1){
        //    //int reqLength = _compute_reqLength(ivalue[0], ivalue[2]);
        //    //if (tidx==0) test[b] = reqLength;
        //    //__syncthreads();                  
        //    //value[tidy*blockDim.x+tidx] = data - medianValue;
        //    //__syncthreads();                  
        //    //_decompress_oneBlock(b*bs*sizeof(float), nb+b*mSize+4, b, reqLength, value, ivalue, cvalue, sums, meta, offsets, midBytes, bi);
        //    bi = false;
        //}

    }

}
