/**************/
/* TIMING GPU */
/**************/

#include "timingGPU.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

struct PrivateTimingGPU {
    hipEvent_t start;
    hipEvent_t stop;
};

// default constructor
TimingGPU::TimingGPU() { privateTimingGPU = new PrivateTimingGPU;  }

// default destructor
TimingGPU::~TimingGPU() { }

void TimingGPU::StartCounter()
{
    hipEventCreate(&((*privateTimingGPU).start));
    hipEventCreate(&((*privateTimingGPU).stop));
    hipEventRecord((*privateTimingGPU).start,0);
}

void TimingGPU::StartCounterFlags()
{
    int eventflags = hipEventBlockingSync;

    hipEventCreateWithFlags(&((*privateTimingGPU).start),eventflags);
    hipEventCreateWithFlags(&((*privateTimingGPU).stop),eventflags);
    hipEventRecord((*privateTimingGPU).start,0);
}

// Gets the counter in ms
float TimingGPU::GetCounter()
{
    float time;
    hipEventRecord((*privateTimingGPU).stop, 0);
    hipEventSynchronize((*privateTimingGPU).stop);
    hipEventElapsedTime(&time,(*privateTimingGPU).start,(*privateTimingGPU).stop);
    return time;
}
