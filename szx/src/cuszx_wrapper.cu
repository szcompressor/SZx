#include "hip/hip_runtime.h"
#include "cuszx_entry.h"
#include "szx_defines.h"
#include "szx_BytesToolkit.h"
#include "szx_TypeManager.h"
#include "timingGPU.h"

extern "C"{
    void cuSZx_integrated_compress(unsigned char *bytes, float *data, float r2r_threshold, float r2r_err, size_t nbEle, int blockSize, size_t *outSize){
        float max,min;
        max = data[0];
        min = data[0];
        for (size_t i = 0; i < nbEle; i++)
        {
            if(data[i] > max) max = data[i];
            if(data[i] < min) min = data[i];
        }
        
        float threshold = r2r_threshold*(max-min);
        float errBound = r2r_err*(max-min);
        bytes = cuSZx_fast_compress_args_unpredictable_blocked_float(data, outSize, errBound, nbEle, blockSize, threshold);
   	printf("outSize %p\n", bytes);
    }

    void cuSZx_integrated_decompress(float *data, unsigned char *bytes, size_t nbEle){
        printf("test\n");
	//cuSZx_fast_decompress_args_unpredictable_blocked_float(&data, nbEle, bytes);
    }

    unsigned char* cuSZx_device_compress(float *oriData, size_t *outSize, float absErrBound, size_t nbEle, int blockSize, float threshold){
        return device_ptr_cuSZx_compress_float(oriData, outSize, absErrBound, nbEle, blockSize, threshold);
    }
    
}
